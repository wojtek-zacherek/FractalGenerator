#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
      printf("%d  %2.2f  %2.2f \n",i,x[i],y[i]);
    //   y[i] = a*x[i] + y[i];
    y[i] = y[2*i] + y[2*i + 1];
  }
}

int main(void)
{
  int N = 1;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(2*N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, 2*N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[2*i] = 2.0f;
    y[2*i+1] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, 2*N*sizeof(float), hipMemcpyHostToDevice);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

  hipMemcpy(y, d_y, 2*N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++){
    maxError = max(maxError, abs(y[i]-4.0f));
    printf("i=%d y=%f\n",i,y[i]);
  }
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}